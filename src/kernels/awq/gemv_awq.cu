#include "hip/hip_runtime.h"
/*
 * Modified from NVIDIA
 * [TRT-LLM](https://github.com/NVIDIA/TensorRT-LLM/tree/d37b507f41a87457fe9f10f7459d08f5db235745/cpp/tensorrt_llm/kernels/weightOnlyBatchedGemv)
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
@article{lin2023awq,
  title={AWQ: Activation-aware Weight Quantization for LLM Compression and Acceleration},
  author={Lin, Ji and Tang, Jiaming and Tang, Haotian and Yang, Shang and Dang, Xingyu and Han, Song},
  journal={arXiv},
  year={2023}
}
*/

#include "gemv_awq.h"
#include "../dispatch_utils.h"

#include "../utils.cuh"
#include "../device_compat.h"
#include <stdio.h>
#include "dequantize.cuh"

#if defined(__HIP_PLATFORM_AMD__)

#include "device_compat.h"

namespace {

constexpr int kInterleave = 4;

__device__ __forceinline__ int compute_ic_div64(int in_features) {
    int value = in_features / 64;
    return value > 0 ? value : 1;
}

__global__ void dequantize_awq_kernel(const int32_t *packed,
                                      const half *scales,
                                      const half *zeros,
                                      half *dequantized,
                                      int rows,
                                      int cols,
                                      int in_features,
                                      int group_size,
                                      int padded_groups,
                                      int scale_stride) {
    const int out_features = rows * kInterleave;
    const int total        = out_features * in_features;
    const int idx          = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) {
        return;
    }

    const int oc_idx = idx / in_features;
    const int ic_idx = idx % in_features;

    const int block     = idx / 32;
    const int offset    = idx % 32;
    const int nibble    = offset / 8;
    const int j         = offset % 8;
    const int index1    = block * 8 + j;
    const int d         = index1 % 16;
    const int tmp       = index1 / 16;
    const int ic_div64  = compute_ic_div64(in_features);
    const int c         = tmp % ic_div64;
    const int row4b     = tmp / ic_div64;
    const int row       = row4b / 4;
    const int b         = row4b % 4;
    const int col_index = ((c * 4 + b) * 16) + d;

    if (row >= rows || col_index >= cols) {
        dequantized[idx] = __float2half(0.0f);
        return;
    }

    const uint16_t packed_val = static_cast<uint16_t>(packed[row * cols + col_index] & 0xFFFF);
    const int nibble_val      = (packed_val >> (4 * nibble)) & 0xF;

    if (padded_groups <= 0) {
        dequantized[idx] = __float2half(0.0f);
        return;
    }

    const int effective_group = group_size > 0 ? group_size : in_features;
    int group_idx              = ic_idx / effective_group;
    if (group_idx < 0) {
        group_idx = 0;
    }
    if (group_idx >= padded_groups) {
        group_idx = padded_groups - 1;
    }

    const float scale = __half2float(scales[group_idx * scale_stride + oc_idx]);
    const float zero  = __half2float(zeros[group_idx * scale_stride + oc_idx]);

    dequantized[idx] = __float2half(nibble_val * scale + zero);
}

__global__ void gemm_fp16_kernel(const half *input,
                                 const half *weight,
                                 half *output,
                                 int M,
                                 int N,
                                 int K) {
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row >= M || col >= N) {
        return;
    }

    float acc = 0.0f;
    for (int k_idx = 0; k_idx < K; ++k_idx) {
        const float a_val = __half2float(input[row * K + k_idx]);
        const float b_val = __half2float(weight[col * K + k_idx]);
        acc += a_val * b_val;
    }

    output[row * N + col] = __float2half(acc);
}

inline void launch_dequantize(const int32_t *packed,
                              const half *scales,
                              const half *zeros,
                              half *dequantized,
                              int rows,
                              int cols,
                              int in_features,
                              int group_size,
                              int padded_groups,
                              int scale_stride) {
    const int out_features = rows * kInterleave;
    const int total        = out_features * in_features;
    constexpr int threads  = 256;
    dim3 block(threads);
    dim3 grid((total + threads - 1) / threads);
    hipLaunchKernelGGL(dequantize_awq_kernel,
                       grid,
                       block,
                       0,
                       getCurrentGpuStream(),
                       packed,
                       scales,
                       zeros,
                       dequantized,
                       rows,
                       cols,
                       in_features,
                       group_size,
                       padded_groups,
                       scale_stride);
    checkCUDA(gpu_runtime::getLastError());
}

inline void launch_gemm(const half *input, const half *weight, half *output, int M, int N, int K) {
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    hipLaunchKernelGGL(gemm_fp16_kernel,
                       grid,
                       block,
                       0,
                       getCurrentGpuStream(),
                       input,
                       weight,
                       output,
                       M,
                       N,
                       K);
    checkCUDA(gpu_runtime::getLastError());
}

} // namespace

Tensor gemv_awq(Tensor _in_feats,
                Tensor _kernel,
                Tensor _scaling_factors,
                Tensor _zeros,
                int m,
                int n,
                int k,
                int group_size) {
    const int rows         = _kernel.size(0);
    const int cols         = _kernel.size(1);
    const int in_features  = k;
    const int out_features = rows * kInterleave;
    const int padded_groups = _scaling_factors.size(0);
    const int scale_stride  = _scaling_factors.size(1);

    assert(out_features == n);

    auto output_shape   = _in_feats.shape.dataExtent;
    output_shape.back() = out_features;

    Tensor dequantized = Tensor::empty({out_features, in_features}, Tensor::FP16, _kernel.device());

    launch_dequantize(_kernel.data_ptr<int32_t>(),
                      _scaling_factors.data_ptr<half>(),
                      _zeros.data_ptr<half>(),
                      dequantized.data_ptr<half>(),
                      rows,
                      cols,
                      in_features,
                      group_size,
                      padded_groups,
                      scale_stride);

    const int computed_m = _in_feats.numel() / in_features;
    assert(computed_m == m);

    Tensor out = Tensor::empty(output_shape, _in_feats.scalarType(), _in_feats.device());
    launch_gemm(_in_feats.data_ptr<half>(),
                dequantized.data_ptr<half>(),
                out.data_ptr<half>(),
                computed_m,
                out_features,
                in_features);

    return out;
}

#else

#define PACK_FACTOR 8
#define WARP_SIZE 32
#define MEM_ACCESS_SIZE 128

// Reduce sum within the warp using the tree reduction algorithm.
template<typename float_t, int Num, int WarpSize>
__device__ __forceinline__ static void warp_reduce(float_t *psum, float (*out_smem)[Num * 4]) {
    // kInterleave = 4
    float fpsum[Num];
#pragma unroll
    for (int i = 0; i < Num; ++i) {
        fpsum[i] = static_cast<float>(psum[i]);
    }

#pragma unroll
    for (int i = 0; i < Num; ++i) {
        // T0 + T1 + T8 + T9 + T16 + T17 + T24 + T25 (kInterleave = 4)
        fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 16);
        fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 8);
        fpsum[i] += __shfl_xor_sync(~0, fpsum[i], 1);
    }
    __syncthreads();
    int warp = threadIdx.x / WarpSize, lane = threadIdx.x % WarpSize;
    if (lane == 0 || lane == 2 || lane == 4 || lane == 6) {
#pragma unroll
        for (int i = 0; i < Num; ++i) {
            out_smem[warp][i * 4 + lane / 2] = fpsum[i];
        }
    }
    __syncthreads();
};

__device__ __forceinline__ int make_divisible(int c, int divisor) {
    return (c + divisor - 1) / divisor;
}

template<typename half_t>
__device__ __forceinline__ packed_as<half_t, 2>::type half2half2(half_t x);

template<>
__device__ __forceinline__ packed_as<half, 2>::type half2half2<half>(half x) {
    return __half2half2(x);
}

template<>
__device__ __forceinline__ packed_as<__hip_bfloat16, 2>::type half2half2<__hip_bfloat16>(__hip_bfloat16 x) {
    return __bfloat162bfloat162(x);
}

template<typename T>
__device__ __forceinline__ float2 half22float2(T val);

template<>
__device__ __forceinline__ float2 half22float2<half2>(half2 val) {
    return __half22float2(val);
}

template<>
__device__ __forceinline__ float2 half22float2<__hip_bfloat162>(__hip_bfloat162 val) {
    return __bfloat1622float2(val);
}

template<typename half_t, int NPerBlock, int Batch, int BlockSize, int GroupSize>
__global__ void gemv_kernel(const half_t *inputs,
                            const uint32_t *weight,
                            const half_t *scales,
                            const half_t *zeros,
                            half_t *outputs,
                            const int IC,
                            const int OC) {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800
    if constexpr (std::is_same_v<half_t, __hip_bfloat16>) {
        trap_unsupported_arch();
        return;
    }
#endif
    using half2_t  = typename packed_as<half_t, 2>::type;
    using accum_t  = float;
    using accum2_t = typename packed_as<accum_t, 2>::type;

    const int kStride            = 64;
    const int kElemsPerThread    = MEM_ACCESS_SIZE / 4;
    const int kThreadsNumPerTile = kStride / kElemsPerThread;
    // assert(MEM_ACCESS_SIZE == 128);

    // static constexpr int kShuffleSize = 32;
    static constexpr int kShuffleBasicTile = 2;
    static constexpr int kShuffleContinous = 4;
    static constexpr int kShuffleStrided   = 4;

    constexpr int Num         = NPerBlock * Batch;
    constexpr int kInterleave = 4;

    alignas(16) half_t local_inputs[kElemsPerThread];
    alignas(16) uint32_t local_qweights[MEM_ACCESS_SIZE / 32];
    alignas(16) half_t half_weight_buffer[kElemsPerThread];
    alignas(16) half_t dequantized_weight[kElemsPerThread * NPerBlock];
    alignas(16) half_t local_scale[NPerBlock];
    alignas(16) half_t local_scaled_zeros[NPerBlock];

    accum_t psum[Num];
    for (int i = 0; i < Num; ++i)
        psum[i] = static_cast<accum_t>(0.f);

    // extern __shared__ uint8_t shmem[];
    // float(*out_smem)[Num * kInterleave] = reinterpret_cast<float(*)[Num * kInterleave]>(shmem);

    __shared__ float out_smem[BlockSize / WARP_SIZE * 2][Num * kInterleave];

    const int blk_row_offset = blockIdx.x * NPerBlock * kInterleave;
    const int thd_row_offset = (threadIdx.x / kThreadsNumPerTile) % kInterleave;
    const int act_k_offset   = threadIdx.x / (kThreadsNumPerTile * kInterleave) * kStride +
                             (threadIdx.x % kThreadsNumPerTile) * kElemsPerThread;
    const int group_offset = act_k_offset / GroupSize;
    // TODO: use make_divisible
    const uint32_t *blk_weight_ptr = weight + blk_row_offset * IC / PACK_FACTOR;
    const half_t *scale_ptr        = scales + blk_row_offset + thd_row_offset + group_offset * OC;
    const half_t *zeros_ptr        = zeros + blk_row_offset + thd_row_offset + group_offset * OC;
    const half_t *inputs_ptr       = inputs + act_k_offset;

    const int act_forward_step   = BlockSize * kElemsPerThread / kInterleave;
    const int scale_forward_step = act_forward_step / GroupSize * OC;

    // Main loop iteration, each block completes the outputs for several OCs
    for (int kk = threadIdx.x * kElemsPerThread; kk < IC * kInterleave; kk += BlockSize * kElemsPerThread) {
// Load qweight, scales and scaled_zeros
#pragma unroll
        for (int idx = 0; idx < NPerBlock; ++idx) {
            // use float4 to load weights, each thread load 32 int4 numbers (1 x float4, 128 bit)
            *((float4 *)(local_qweights)) = *((float4 *)(blk_weight_ptr + (idx * kInterleave * IC + kk) / PACK_FACTOR));
            local_scale[idx]              = *(scale_ptr + idx * kInterleave);
            local_scaled_zeros[idx]       = *(zeros_ptr + idx * kInterleave);

// Map int4 qweight to fp format
#pragma unroll
            for (int i = 0; i < MEM_ACCESS_SIZE / 32; ++i) {
                // Converts 32 bits (8 x int4) to 8 fp16
                dequantize_s4_to_fp16x2(*reinterpret_cast<half2_t *>(local_qweights + i),
                                        reinterpret_cast<uint4 *>(half_weight_buffer + i * PACK_FACTOR));
            }

// Dequantize (apply s/z) and shuffle elements to match the weight packing format
#pragma unroll
            for (int i = 0; i < kShuffleContinous; ++i) {
#pragma unroll
                for (int j = 0; j < kShuffleStrided; ++j) {
                    half2_t w = *reinterpret_cast<half2_t *>(half_weight_buffer +
                                                             (i + j * kShuffleContinous) * kShuffleBasicTile);
                    w         = __hfma2(w, half2half2(local_scale[idx]), half2half2(local_scaled_zeros[idx]));
                    dequantized_weight[((i * kShuffleStrided + j) * kShuffleBasicTile + 0) * NPerBlock + idx] = w.x;
                    dequantized_weight[((i * kShuffleStrided + j) * kShuffleBasicTile + 1) * NPerBlock + idx] = w.y;
                }
            }
        }
#pragma unroll
        for (int batch_idx = 0; batch_idx < Batch; ++batch_idx) {
            const half_t *local_inputs_ptr = inputs_ptr + batch_idx * IC;
#pragma unroll
            for (int idx = 0; idx < kElemsPerThread / 8; ++idx) {
                // load activation, 8 halves (128 bits) / step.
                *((float4 *)(local_inputs + idx * 8)) = *((float4 *)(local_inputs_ptr + idx * 8));
            }
// Perform the MACs
#pragma unroll
            for (int x = 0; x < NPerBlock / 2; ++x) {
#pragma unroll
                for (int y = 0; y < kElemsPerThread; ++y) {
                    accum2_t prod = cuda_cast<accum2_t>(
                        __hmul2(*reinterpret_cast<half2_t *>(dequantized_weight + y * NPerBlock + x * 2),
                                half2half2(local_inputs[y])));
                    *reinterpret_cast<accum2_t *>(psum + batch_idx * NPerBlock + x * 2) =
                        prod + *reinterpret_cast<accum2_t *>(psum + batch_idx * NPerBlock + x * 2);
                    // *reinterpret_cast<half2_t*>(psum + batch_idx * NPerBlock + x * 2)
                    //     = __hfma2(*reinterpret_cast<half2_t*>(dequantized_weight + y * NPerBlock + x * 2),
                    //         half2half2(local_inputs[y]),
                    //         *reinterpret_cast<half2_t*>(psum + batch_idx * NPerBlock + x * 2));
                }
            }
        }
        inputs_ptr += act_forward_step;
        scale_ptr += scale_forward_step;
        zeros_ptr += scale_forward_step;
    }

    warp_reduce<accum_t, Num, WARP_SIZE>(psum, out_smem);

    // Num * Interleave = batch * NPerBlock * Interleave -> 1 thread_block write back num
    for (int i = threadIdx.x; i < Num * kInterleave; i += BlockSize) {
        int batch_idx = i / (NPerBlock * kInterleave);
        int oc_idx    = i % (NPerBlock * kInterleave);
        float acc     = 0.f;
        for (int j = 0; j < BlockSize / WARP_SIZE; ++j) {
            acc += out_smem[j][i];
        }
        outputs[batch_idx * OC + blk_row_offset + oc_idx] = static_cast<half_t>(acc);
    }
}

/*
Computes GEMV (PyTorch interface).

Args:
  _in_feats: tensor of shape [B, IC];
  _kernel: int tensor of shape [OC, IC // 8];
  _zeros: int tensor of shape [OC, IC // G // 8];
  _scaling_factors: tensor of shape [OC, IC // G];
  blockDim_x: size of thread block, dimension x, where blockDim_x * workload_per_thread = IC;
  blockDim_y: size of thread block, dimension y, where blockDim_y * gridDim_y = OC;

Returns:
  out_feats: tensor of shape [B, OC];
*/
Tensor gemv_awq(
    Tensor _in_feats, Tensor _kernel, Tensor _scaling_factors, Tensor _zeros, int m, int n, int k, int group_size) {
    return dispatchFloat16(_scaling_factors.scalar_type(), [&]<typename half_t>() {
        assert(isTypeMatch<half_t>(_in_feats.dtype()));

        auto output_shape   = _in_feats.shape.dataExtent;
        output_shape.back() = n;

        auto in_feats        = reinterpret_cast<half_t *>(_in_feats.data_ptr<half_t>());
        auto kernel          = reinterpret_cast<uint32_t *>(_kernel.data_ptr());
        auto zeros           = reinterpret_cast<half_t *>(_zeros.data_ptr<half_t>());
        auto scaling_factors = reinterpret_cast<half_t *>(_scaling_factors.data_ptr<half_t>());

        Tensor _out_feats = Tensor::allocate(output_shape, _in_feats.dtype(), _in_feats.device());
        half_t *out_feats = reinterpret_cast<half_t *>(_out_feats.data_ptr());

        static constexpr int N_PER_BLOCK  = 2;
        static constexpr int K_INTERLEAVE = 4;
        static constexpr int BLOCK_SIZE   = 256;

        dim3 num_blocks(n / N_PER_BLOCK / K_INTERLEAVE);
        dim3 num_threads(BLOCK_SIZE);

        constexpr int GROUP_SIZE = 64;

        assert(m > 0 && m <= 8);
        assert(group_size == GROUP_SIZE);

        dispatchVal(m, std::make_integer_sequence<int, 9>(), [&]<int M>() {
            if constexpr (M == 0) {
                assert(false);
                return;
            }
            if constexpr (M > 0) {
                gemv_kernel<half_t, N_PER_BLOCK, M, BLOCK_SIZE, GROUP_SIZE>
                    <<<num_blocks, num_threads, 0, getCurrentGpuStream()>>>(
                        in_feats, kernel, scaling_factors, zeros, out_feats, k, n);
                checkCUDA(gpu_runtime::getLastError());
            }
        });

        return _out_feats;
    });
}

#endif // defined(__HIP_PLATFORM_AMD__)
